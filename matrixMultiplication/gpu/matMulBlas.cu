#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdlib>
#include <ctime>

void checkCudaError(hipError_t status, const char *msg) {
    if (status != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t status, const char *msg) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << msg << ": CUBLAS error" << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 3072; // Matrix size N x N
    const size_t matrixSize = N * N * sizeof(double);

    // Host matrices
    double *h_A = new double[N * N];
    double *h_B = new double[N * N];
    double *h_C = new double[N * N];

    // Initialize host matrices with random values
    srand(static_cast<unsigned>(time(nullptr)));
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<double>(rand()) / RAND_MAX;
        h_B[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    // Device matrices
    double *d_A, *d_B, *d_C;

    // Allocate device memory
    checkCudaError(hipMalloc((void **)&d_A, matrixSize), "Failed to allocate device memory for A");
    checkCudaError(hipMalloc((void **)&d_B, matrixSize), "Failed to allocate device memory for B");
    checkCudaError(hipMalloc((void **)&d_C, matrixSize), "Failed to allocate device memory for C");

    // Copy matrices to device
    checkCudaError(hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice), "Failed to copy A to device");
    checkCudaError(hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice), "Failed to copy B to device");

    // cuBLAS handle
    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle), "Failed to create cuBLAS handle");

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    const double alpha = 1.0;
    const double beta = 0.0;

    checkCublasError(
        hipblasDgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,  // Transpose options
            N, N, N,                  // Dimensions
            &alpha,                   // Scaling factor alpha
            d_A, N,                   // Matrix A
            d_B, N,                   // Matrix B
            &beta,                    // Scaling factor beta
            d_C, N                    // Result matrix C
        ),
        "Failed to perform DGEMM operation"
    );

    // Copy result back to host
    checkCudaError(hipMemcpy(h_C, d_C, matrixSize, hipMemcpyDeviceToHost), "Failed to copy C to host");

    // Print a small portion of the result to verify correctness
    std::cout << "Result matrix C (first 5x5 block):" << std::endl;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}

